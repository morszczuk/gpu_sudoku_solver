#include "sudoku_kernel.h"

void cudaErrorHandling(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error on CUDA %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
	}
}

void displayHostArray(char* title, int* array, int N, int M)
{
  printf("---------%s-----------\n", title);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
			printf("%d |", array[i*N + j]);
		
		printf("\n");

		for (int j = 0; j < N; j++)
			printf("- |");
		
		printf("\n");
	}
  printf("------------------------------\n");
}

int* copySudokuToDevice(int* h_sudoku)
{
	int* d_sudoku;
	
	cudaErrorHandling(hipMalloc((void **)&d_sudoku, NN * NN * sizeof(int)));

	cudaErrorHandling(hipMemcpy(d_sudoku, h_sudoku, NN * NN * sizeof(int), hipMemcpyHostToDevice));

	return d_sudoku;
}

int* copySudokuToHost(int* d_sudoku)
{
	int* h_sudoku = new int[NN*NN];

	cudaErrorHandling(hipMemcpy(h_sudoku, d_sudoku, NN * NN * sizeof(int), hipMemcpyDeviceToHost));

	return h_sudoku;
}

resolution* createRowSolution(int row, int* current_solution)
{
	int *d_current_solution = copySudokuToDevice(current_solution);
	int *lalala = copySudokuToHost(d_current_solution);
	displayHostArray("CREATE ROW SOLUTION", lalala, NN, NN);

}

hipError_t solveSudoku(int* h_sudoku_solved, int* h_sudoku_unsolved)
{
  int* resolution = new int [NN*NN];
  displayHostArray("RESOLUTION", resolution, NN, NN);

	createRowSolution(0, resolution);
}

