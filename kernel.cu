#include "hip/hip_runtime.h"
#include "kernel.h"

void cudaErrorHandling(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error on CUDA %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
	}
}

__global__ void checkQuizFill(int d_quiz[SUD_SIZE][SUD_SIZE], int d_fill)
{
	int idx = blockDim.y*blockIdx.y + threadIdx.y;
	int idy = blockDim.x*blockIdx.x + threadIdx.x;

	//] = d_quiz[idx][idy] > 0 ? 1 : 0;
}

__global__ void checkCorrectness(int* d_sudoku, int* d_number_presence)
{
	extern __shared__ int number_presence[];
	int idx = blockDim.y*blockIdx.y + threadIdx.y;
	int idy = blockDim.x*blockIdx.x + threadIdx.x;
	int index_1, index_2, index_3;
	int k = 81;

	number_presence[idx * 9 + idy] = 0;
	number_presence[k + idx * 9 + idy] = 0;
	number_presence[(2*k) + (idx * 9 + idy)] = 0;

	index_1 = idx * 9 + d_sudoku[idx/SUD_SIZE + idy] - 1;
	index_2 = k + idy * 9 + d_sudoku[idx/SUD_SIZE + idy] - 1;
	index_3 = (2 * k) + ((idx / 3) * 27) + ((idy / 3) * 9) + d_sudoku[idx/SUD_SIZE + idy] - 1;

	printf("[idx: %d, idy: %d | val: %d | %d, %d, %d]\n", idx, idy, d_sudoku[idx/SUD_SIZE + idy], index_1, index_2 - k , index_3 - (2*k));
}

hipError_t solveSudoku(int* h_sudoku_quiz)
{
	int *d_sudoku_quiz, *d_quiz_fill, *d_number_presence;
	int sharedMemorySize;
	cudaErrorHandling(hipMalloc((void **)&d_sudoku_quiz, SUD_SIZE * SUD_SIZE * sizeof(int)));
	cudaErrorHandling(hipMalloc((void **)&d_quiz_fill, SUD_SIZE * SUD_SIZE * sizeof(int)));

	cudaErrorHandling(hipMemcpy(d_sudoku_quiz, h_sudoku_quiz, SUD_SIZE * SUD_SIZE * sizeof(int), hipMemcpyHostToDevice));

	cudaErrorHandling(hipMalloc((void **)&d_number_presence, 243 * sizeof(int)));

	dim3 dimBlock = dim3(9, 9, 1);
	dim3 dimGrid = dim3(1);
	sharedMemorySize = 243 * sizeof(int);
	checkCorrectness <<<dimGrid, dimBlock, sharedMemorySize>>> (d_sudoku_quiz, d_number_presence);
	cudaErrorHandling(hipDeviceSynchronize());
	//int h_sudoku_quiz[SUD_SIZE][SUD_SIZE];

	//for(int i = 0; i < SUD_SIZE; i++)
	//	for (int j = 0; j < SUD_SIZE; j++)
	//		h_sudoku_quiz[i][j] = _sudoku_quiz[i][j];
}