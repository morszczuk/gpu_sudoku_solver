#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <string>
#include <sstream>
#include <time.h>
#include "constants.h"
#include "sudoku_parser.h"

//reading sudoku quiz from a file
// int* readSudokuArray(char* filename)
// {
// 	int* h_sudoku = new int[SUD_SIZE*SUD_SIZE];

// 	//printf("SUDOKU FILENAME: %s\n", filename);
// 	std::ifstream sudoku_file(filename);

// 	int a0, a1, a2, a3, a4, a5, a6, a7, a8;
// 	int i = 0;

// 	while (sudoku_file >> a0 >> a1 >> a2 >> a3 >> a4 >> a5 >> a6 >> a7 >> a8)
// 	{
// 		h_sudoku[i + 0] = a0;
// 		h_sudoku[i + 1] = a1;
// 		h_sudoku[i + 2] = a2;
// 		h_sudoku[i + 3] = a3;
// 		h_sudoku[i + 4] = a4;
// 		h_sudoku[i + 5] = a5;
// 		h_sudoku[i + 6] = a6;
// 		h_sudoku[i + 7] = a7;
// 		h_sudoku[i + 8] = a8;
// 		i++;
// 	}

// 	return h_sudoku;
// }

// //printing Array in sudoku-style.
// void printArray(int* array, int N, int M)
// {
// 	for (int i = 0; i < N; i++)
// 	{
// 		for (int j = 0; j < M; j++)
// 			printf("%d |", array[i/N + j]);
		
// 		printf("\n");

// 		for (int j = 0; j < N; j++)
// 			printf("- |");
		
// 		printf("\n");
// 	}
// }

__global__ void checkQuizFill(int d_quiz[SUD_SIZE][SUD_SIZE], int d_fill)
{
	int idx = blockDim.y*blockIdx.y + threadIdx.y;
	int idy = blockDim.x*blockIdx.x + threadIdx.x;

	//] = d_quiz[idx][idy] > 0 ? 1 : 0;
}

__global__ void checkCorrectness(int* d_sudoku, int* d_number_presence)
{
	fprintf(stdout, "madafaka3\n");
	extern __shared__ int number_presence[];
	int idx = blockDim.y*blockIdx.y + threadIdx.y;
	int idy = blockDim.x*blockIdx.x + threadIdx.x;
	printf("idx: %d", idx);
	printf("idy: %d", idy);
}

hipError_t solveSudoku(int* h_sudoku_quiz)
{
	int *d_sudoku_quiz, *d_quiz_fill, *d_number_presence;
	int sharedMemorySize;
	int f;
	printf("madafaka\n");
	hipMalloc((void **)&d_sudoku_quiz, SUD_SIZE * SUD_SIZE * sizeof(int));
	hipMalloc((void **)&d_quiz_fill, SUD_SIZE * SUD_SIZE * sizeof(int));

	hipMemcpy(d_sudoku_quiz, h_sudoku_quiz, SUD_SIZE * SUD_SIZE * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **)&d_number_presence, 243 * sizeof(int));
	printf("madafaka2\n");

	dim3 dimBlock = dim3(9, 9, 1);
	dim3 dimGrid = dim3(1);
	sharedMemorySize = 243 * sizeof(int);
	fprintf(stdout, "madafaka3\n");
	checkCorrectness <<<dimGrid, dimBlock, sharedMemorySize>>> (d_sudoku_quiz, d_number_presence);
	hipDeviceSynchronize();
	//int h_sudoku_quiz[SUD_SIZE][SUD_SIZE];

	//for(int i = 0; i < SUD_SIZE; i++)
	//	for (int j = 0; j < SUD_SIZE; j++)
	//		h_sudoku_quiz[i][j] = _sudoku_quiz[i][j];
}

int main()
{
	char filename[] = "quizzes/arr_1_solved.txt";
	int * h_sudoku_quiz;
	int a =5;
	
	//RETRIEVING SUDOKU QUIZ
	h_sudoku_quiz = readSudokuArray(filename);
	printArray(h_sudoku_quiz, SUD_SIZE, SUD_SIZE);

	//STARTING TIME MEASURMENT
	clock_t begin = clock();
	
	//SOLVING SUDOKU 
	hipError_t cudaStatus = solveSudoku(h_sudoku_quiz);
	if (cudaStatus != hipSuccess) {
		printf("fds");
		fprintf(stderr, "solveSudoku failed!");
		return 1;
	}

	//ENDING TIME MEASURMENT
	clock_t end = clock();
	printf("[FUNCTION TIME] %f ms\n", (double)(end - begin) / CLOCKS_PER_SEC * 1000);


	getchar();

	// RESETING CUDA DEVICE
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}